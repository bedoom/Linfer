#include "hip/hip_runtime.h"
#include "trt_common/infer.hpp"
#include "yolov8.hpp"
#include "trt_common/preprocess_kernel.cuh"
#include "trt_common/cuda_tools.hpp"
#include "trt_common/ilogger.hpp"

namespace yolo {

using namespace std;

#define GPU_BLOCK_THREADS 512
#define checkRuntime(call)                                                                 \
  do {                                                                                     \
    auto ___call__ret_code__ = (call);                                                     \
    if (___call__ret_code__ != hipSuccess) {                                              \
      INFO("CUDA Runtime error💥 %s # %s, code = %s [ %d ]", #call,                         \
           hipGetErrorString(___call__ret_code__), hipGetErrorName(___call__ret_code__), \
           ___call__ret_code__);                                                           \
      abort();                                                                             \
    }                                                                                      \
  } while (0)

#define checkKernel(...)                 \
  do {                                   \
    { (__VA_ARGS__); }                   \
    checkRuntime(hipPeekAtLastError()); \
  } while (0)


const int NUM_BOX_ELEMENT = 8;  // left, top, right, bottom, confidence, class,
                                // keepflag, row_index(output)
const int MAX_IMAGE_BOXES = 1024;
inline int upbound(int n, int align = 32) { return (n + align - 1) / align * align; }
static __host__ __device__ void affine_project(float *matrix, float x, float y, float *ox,
                                               float *oy) {
  *ox = matrix[0] * x + matrix[1] * y + matrix[2];
  *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

static __global__ void decode_kernel_common(float *predict, int num_bboxes, int num_classes,
                                            int output_cdim, float confidence_threshold,
                                            float *invert_affine_matrix, float *parray,
                                            int MAX_IMAGE_BOXES) {
  int position = blockDim.x * blockIdx.x + threadIdx.x;
  if (position >= num_bboxes) return;

  float *pitem = predict + output_cdim * position;
  float objectness = pitem[4];
  if (objectness < confidence_threshold) return;

  float *class_confidence = pitem + 5;
  float confidence = *class_confidence++;
  int label = 0;
  for (int i = 1; i < num_classes; ++i, ++class_confidence) {
    if (*class_confidence > confidence) {
      confidence = *class_confidence;
      label = i;
    }
  }

  confidence *= objectness;
  if (confidence < confidence_threshold) return;

  int index = atomicAdd(parray, 1);
  if (index >= MAX_IMAGE_BOXES) return;

  float cx = *pitem++;
  float cy = *pitem++;
  float width = *pitem++;
  float height = *pitem++;
  float left = cx - width * 0.5f;
  float top = cy - height * 0.5f;
  float right = cx + width * 0.5f;
  float bottom = cy + height * 0.5f;
  affine_project(invert_affine_matrix, left, top, &left, &top);
  affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

  float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
  *pout_item++ = left;
  *pout_item++ = top;
  *pout_item++ = right;
  *pout_item++ = bottom;
  *pout_item++ = confidence;
  *pout_item++ = label;
  *pout_item++ = 1;  // 1 = keep, 0 = ignore
}
/// ------------------ 核函数定义 ------------------
static __global__ void decode_kernel_v8(float *predict, int num_bboxes, int num_classes,
                                        int output_cdim, float confidence_threshold,
                                        float *invert_affine_matrix, float *parray,
                                        int MAX_IMAGE_BOXES) {
  int position = blockDim.x * blockIdx.x + threadIdx.x;
  if (position >= num_bboxes) return;

  float *pitem = predict + output_cdim * position;
  float *class_confidence = pitem + 4;
  float confidence = *class_confidence++;
  int label = 0;
  for (int i = 1; i < num_classes; ++i, ++class_confidence) {
    if (*class_confidence > confidence) {
      confidence = *class_confidence;
      label = i;
    }
  }
  if (confidence < confidence_threshold) return;

  int index = atomicAdd(parray, 1);
  if (index >= MAX_IMAGE_BOXES) return;

  float cx = *pitem++;
  float cy = *pitem++;
  float width = *pitem++;
  float height = *pitem++;
  float left = cx - width * 0.5f;
  float top = cy - height * 0.5f;
  float right = cx + width * 0.5f;
  float bottom = cy + height * 0.5f;
  affine_project(invert_affine_matrix, left, top, &left, &top);
  affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

  float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
  *pout_item++ = left;
  *pout_item++ = top;
  *pout_item++ = right;
  *pout_item++ = bottom;
  *pout_item++ = confidence;
  *pout_item++ = label;
  *pout_item++ = 1;  // 1 = keep, 0 = ignore
  *pout_item++ = position;
}

static __device__ float box_iou(float aleft, float atop, float aright, float abottom, float bleft,
                                float btop, float bright, float bbottom) {
  float cleft = max(aleft, bleft);
  float ctop = max(atop, btop);
  float cright = min(aright, bright);
  float cbottom = min(abottom, bbottom);

  float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
  if (c_area == 0.0f) return 0.0f;

  float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
  float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
  return c_area / (a_area + b_area - c_area);
}

static __global__ void fast_nms_kernel(float *bboxes, int MAX_IMAGE_BOXES, float threshold) {
  int position = (blockDim.x * blockIdx.x + threadIdx.x);
  int count = min((int)*bboxes, MAX_IMAGE_BOXES);
  if (position >= count) return;

  // left, top, right, bottom, confidence, class, keepflag
  float *pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
  for (int i = 0; i < count; ++i) {
    float *pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
    if (i == position || pcurrent[5] != pitem[5]) continue;

    if (pitem[4] >= pcurrent[4]) {
      if (pitem[4] == pcurrent[4] && i < position) continue;

      float iou = box_iou(pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3], pitem[0], pitem[1],
                          pitem[2], pitem[3]);

      if (iou > threshold) {
        pcurrent[6] = 0;  // 1=keep, 0=ignore
        return;
      }
    }
  }
}


static void decode_kernel_invoker(float *predict, int num_bboxes, int num_classes, int output_cdim,
                                  float confidence_threshold, float nms_threshold,
                                  float *invert_affine_matrix, float *parray, int MAX_IMAGE_BOXES,
                                  Type type, hipStream_t stream) {
  auto grid = CUDATools::grid_dims(num_bboxes);
  auto block = CUDATools::block_dims(num_bboxes);

  if (type == Type::V8) {
    checkKernel(decode_kernel_v8<<<grid, block, 0, stream>>>(
        predict, num_bboxes, num_classes, output_cdim, confidence_threshold, invert_affine_matrix,
        parray, MAX_IMAGE_BOXES));
  } else {
    checkKernel(decode_kernel_common<<<grid, block, 0, stream>>>(
        predict, num_bboxes, num_classes, output_cdim, confidence_threshold, invert_affine_matrix,
        parray, MAX_IMAGE_BOXES));
  }

  grid = CUDATools::grid_dims(MAX_IMAGE_BOXES);
  block = CUDATools::block_dims(MAX_IMAGE_BOXES);
  checkKernel(fast_nms_kernel<<<grid, block, 0, stream>>>(parray, MAX_IMAGE_BOXES, nms_threshold));
}


static __global__ void decode_single_mask_kernel(int left, int top, float *mask_weights,
                                                 float *mask_predict, int mask_width,
                                                 int mask_height, unsigned char *mask_out,
                                                 int mask_dim, int out_width, int out_height) {
  // mask_predict to mask_out
  // mask_weights @ mask_predict
  int dx = blockDim.x * blockIdx.x + threadIdx.x;
  int dy = blockDim.y * blockIdx.y + threadIdx.y;
  if (dx >= out_width || dy >= out_height) return;

  int sx = left + dx;
  int sy = top + dy;
  if (sx < 0 || sx >= mask_width || sy < 0 || sy >= mask_height) {
    mask_out[dy * out_width + dx] = 0;
    return;
  }

  float cumprod = 0;
  for (int ic = 0; ic < mask_dim; ++ic) {
    float cval = mask_predict[(ic * mask_height + sy) * mask_width + sx];
    float wval = mask_weights[ic];
    cumprod += cval * wval;
  }

  float alpha = 1.0f / (1.0f + exp(-cumprod));
  mask_out[dy * out_width + dx] = alpha * 255;
}

static void decode_single_mask(float left, float top, float *mask_weights, float *mask_predict,
                               int mask_width, int mask_height, unsigned char *mask_out,
                               int mask_dim, int out_width, int out_height, hipStream_t stream) {
  // mask_weights is mask_dim(32 element) gpu pointer
  dim3 grid((out_width + 31) / 32, (out_height + 31) / 32);
  dim3 block(32, 32);

  checkKernel(decode_single_mask_kernel<<<grid, block, 0, stream>>>(
      left, top, mask_weights, mask_predict, mask_width, mask_height, mask_out, mask_dim, out_width,
      out_height));
}

const char *type_name(Type type) {
  switch (type) {
    case Type::V5:
      return "YoloV5";
    case Type::V3:
      return "YoloV3";
    case Type::V7:
      return "YoloV7";
    case Type::X:
      return "YoloX";
    case Type::V8:
      return "YoloV8";
    default:
      return "Unknow";
  }
}

struct AffineMatrix {
  float i2d[6];  // image to dst(network), 2x3 matrix
  float d2i[6];  // dst to image, 2x3 matrix

  void compute(const std::tuple<int, int> &from, const std::tuple<int, int> &to) {
    float scale_x = get<0>(to) / (float)get<0>(from);
    float scale_y = get<1>(to) / (float)get<1>(from);
    float scale = std::min(scale_x, scale_y);
    i2d[0] = scale;
    i2d[1] = 0;
    i2d[2] = -scale * get<0>(from) * 0.5 + get<0>(to) * 0.5 + scale * 0.5 - 0.5;
    i2d[3] = 0;
    i2d[4] = scale;
    i2d[5] = -scale * get<1>(from) * 0.5 + get<1>(to) * 0.5 + scale * 0.5 - 0.5;

    double D = i2d[0] * i2d[4] - i2d[1] * i2d[3];
    D = D != 0. ? double(1.) / D : double(0.);
    double A11 = i2d[4] * D, A22 = i2d[0] * D, A12 = -i2d[1] * D, A21 = -i2d[3] * D;
    double b1 = -A11 * i2d[2] - A12 * i2d[5];
    double b2 = -A21 * i2d[2] - A22 * i2d[5];

    d2i[0] = A11;
    d2i[1] = A12;
    d2i[2] = b1;
    d2i[3] = A21;
    d2i[4] = A22;
    d2i[5] = b2;
  }
};

InstanceSegmentMap::InstanceSegmentMap(int width, int height) {
  this->width = width;
  this->height = height;
  checkRuntime(hipHostMalloc(&this->data, width * height));
}

InstanceSegmentMap::~InstanceSegmentMap() {
  if (this->data) {
    checkRuntime(hipHostFree(this->data));
    this->data = nullptr;
  }
  this->width = 0;
  this->height = 0;
}

class InferImpl : public Infer {
 public:
  shared_ptr<trt::Infer> trt_;
  string engine_file_;
  Type type_;
  float confidence_threshold_;
  float nms_threshold_;
  vector<shared_ptr<trt::Memory<unsigned char>>> preprocess_buffers_;
  trt::Memory<float> input_buffer_, bbox_predict_, output_boxarray_;
  trt::Memory<float> segment_predict_;
  int network_input_width_, network_input_height_;
  CUDAKernel::Norm normalize_;
  vector<int> bbox_head_dims_;
  vector<int> segment_head_dims_;
  int num_classes_ = 0;
  // bool has_segment_ = false;
  bool isdynamic_model_ = false;
  vector<shared_ptr<trt::Memory<unsigned char>>> box_segment_cache_;

  virtual ~InferImpl() = default;

  void adjust_memory(int batch_size) {
    // the inference batch_size
    std::size_t input_numel = network_input_width_ * network_input_height_ * 3;
    input_buffer_.gpu(batch_size * input_numel);
    bbox_predict_.gpu(batch_size * bbox_head_dims_[1] * bbox_head_dims_[2]);
    output_boxarray_.gpu(batch_size * (32 + MAX_IMAGE_BOXES * NUM_BOX_ELEMENT));
    output_boxarray_.cpu(batch_size * (32 + MAX_IMAGE_BOXES * NUM_BOX_ELEMENT));

    if ((int)preprocess_buffers_.size() < batch_size) {
      for (int i = preprocess_buffers_.size(); i < batch_size; ++i)
        preprocess_buffers_.push_back(make_shared<trt::Memory<unsigned char>>());
    }
  }

  void preprocess(int ibatch, const Image &image,
                  shared_ptr<trt::Memory<unsigned char>> preprocess_buffer, AffineMatrix &affine,
                  void *stream = nullptr) {
    affine.compute(make_tuple(image.width, image.height),
                   make_tuple(network_input_width_, network_input_height_));

    std::size_t input_numel = network_input_width_ * network_input_height_ * 3;
    float *input_device = input_buffer_.gpu() + ibatch * input_numel;
    std::size_t size_image = image.width * image.height * 3;
    std::size_t size_matrix = upbound(sizeof(affine.d2i), 32);
    uint8_t *gpu_workspace = preprocess_buffer->gpu(size_matrix + size_image);
    float *affine_matrix_device = (float *)gpu_workspace;
    uint8_t *image_device = gpu_workspace + size_matrix;

    uint8_t *cpu_workspace = preprocess_buffer->cpu(size_matrix + size_image);
    float *affine_matrix_host = (float *)cpu_workspace;
    uint8_t *image_host = cpu_workspace + size_matrix;

    // speed up
    hipStream_t stream_ = (hipStream_t)stream;
    memcpy(image_host, image.bgrptr, size_image);
    memcpy(affine_matrix_host, affine.d2i, sizeof(affine.d2i));
    checkRuntime(
        hipMemcpyAsync(image_device, image_host, size_image, hipMemcpyHostToDevice, stream_));
    checkRuntime(hipMemcpyAsync(affine_matrix_device, affine_matrix_host, sizeof(affine.d2i),
                                 hipMemcpyHostToDevice, stream_));

    warp_affine_bilinear_and_normalize_plane(image_device, image.width * 3, image.width,
                                             image.height, input_device, network_input_width_,
                                             network_input_height_, affine_matrix_device, 114,
                                             normalize_, stream_);
  }

  bool load(const string &engine_file, Type type, float confidence_threshold, float nms_threshold) {
    trt_ = trt::load(engine_file);
    if (trt_ == nullptr) return false;

    trt_->print();

    this->type_ = type;
    this->confidence_threshold_ = confidence_threshold;
    this->nms_threshold_ = nms_threshold;

    auto input_dim = trt_->static_dims(0);
    bbox_head_dims_ = trt_->static_dims(1);
    network_input_width_ = input_dim[3];
    network_input_height_ = input_dim[2];
    isdynamic_model_ = trt_->has_dynamic_dim();

    if (type == Type::V5 || type == Type::V3 || type == Type::V7) {
      normalize_ = CUDAKernel::Norm::alpha_beta(1 / 255.0f, 0.0f, CUDAKernel::ChannelType::Invert);
      num_classes_ = bbox_head_dims_[2] - 5;
    } else if (type == Type::V8) {
      normalize_ = CUDAKernel::Norm::alpha_beta(1 / 255.0f, 0.0f, CUDAKernel::ChannelType::Invert);
      num_classes_ = bbox_head_dims_[2] - 4;
    } else if (type == Type::V8Seg) {
      normalize_ = CUDAKernel::Norm::alpha_beta(1 / 255.0f, 0.0f, CUDAKernel::ChannelType::Invert);
      num_classes_ = bbox_head_dims_[2] - 4 - segment_head_dims_[1];
    } else if (type == Type::X) {
      normalize_ = CUDAKernel::Norm::None();
      num_classes_ = bbox_head_dims_[2] - 5;
    } else {
      INFO("Unsupport type %d", type);
    }
    return true;
  }

  virtual BoxArray forward(const Image &image, void *stream = nullptr) override {
    auto output = forwards({image}, stream);
    if (output.empty()) return {};
    return output[0];
  }

  virtual vector<BoxArray> forwards(const vector<Image> &images, void *stream = nullptr) override {
    int num_image = images.size();
    if (num_image == 0) return {};

    auto input_dims = trt_->static_dims(0);
    int infer_batch_size = input_dims[0];
    if (infer_batch_size != num_image) {
      if (isdynamic_model_) {
        infer_batch_size = num_image;
        input_dims[0] = num_image;
        if (!trt_->set_run_dims(0, input_dims)) return {};
      } else {
        if (infer_batch_size < num_image) {
          INFO(
              "When using static shape model, number of images[%d] must be "
              "less than or equal to the maximum batch[%d].",
              num_image, infer_batch_size);
          return {};
        }
      }
    }
    adjust_memory(infer_batch_size);

    vector<AffineMatrix> affine_matrixs(num_image);
    hipStream_t stream_ = (hipStream_t)stream;
    for (int i = 0; i < num_image; ++i)
      preprocess(i, images[i], preprocess_buffers_[i], affine_matrixs[i], stream);

    float *bbox_output_device = bbox_predict_.gpu();
    vector<void *> bindings{input_buffer_.gpu(), bbox_output_device};

    if (!trt_->forward(bindings, stream)) {
      INFO("Failed to tensorRT forward.");
      return {};
    }

    for (int ib = 0; ib < num_image; ++ib) {
      float *boxarray_device =
          output_boxarray_.gpu() + ib * (32 + MAX_IMAGE_BOXES * NUM_BOX_ELEMENT);
      float *affine_matrix_device = (float *)preprocess_buffers_[ib]->gpu();
      float *image_based_bbox_output =
          bbox_output_device + ib * (bbox_head_dims_[1] * bbox_head_dims_[2]);
      checkRuntime(hipMemsetAsync(boxarray_device, 0, sizeof(int), stream_));
      decode_kernel_invoker(image_based_bbox_output, bbox_head_dims_[1], num_classes_,
                            bbox_head_dims_[2], confidence_threshold_, nms_threshold_,
                            affine_matrix_device, boxarray_device, MAX_IMAGE_BOXES, type_, stream_);
    }
    checkRuntime(hipMemcpyAsync(output_boxarray_.cpu(), output_boxarray_.gpu(),
                                 output_boxarray_.gpu_bytes(), hipMemcpyDeviceToHost, stream_));
    checkRuntime(hipStreamSynchronize(stream_));

    vector<BoxArray> arrout(num_image);
    int imemory = 0;
    for (int ib = 0; ib < num_image; ++ib) {
      float *parray = output_boxarray_.cpu() + ib * (32 + MAX_IMAGE_BOXES * NUM_BOX_ELEMENT);
      int count = min(MAX_IMAGE_BOXES, (int)*parray);
      BoxArray &output = arrout[ib];
      output.reserve(count);
      for (int i = 0; i < count; ++i) {
        float *pbox = parray + 1 + i * NUM_BOX_ELEMENT;
        int label = pbox[5];
        int keepflag = pbox[6];
        if (keepflag == 1) {
          Box result_object_box(pbox[0], pbox[1], pbox[2], pbox[3], pbox[4], label);
          output.emplace_back(result_object_box);
        }
      }
    }

    return arrout;
  }
};

Infer *loadraw(const std::string &engine_file, Type type, float confidence_threshold,
               float nms_threshold) {
  InferImpl *impl = new InferImpl();
  if (!impl->load(engine_file, type, confidence_threshold, nms_threshold)) {
    delete impl;
    impl = nullptr;
  }
  return impl;
}

shared_ptr<Infer> load(const string &engine_file, Type type, float confidence_threshold,
                       float nms_threshold) {
  return std::shared_ptr<InferImpl>(
      (InferImpl *)loadraw(engine_file, type, confidence_threshold, nms_threshold));
}

std::tuple<uint8_t, uint8_t, uint8_t> hsv2bgr(float h, float s, float v) {
  const int h_i = static_cast<int>(h * 6);
  const float f = h * 6 - h_i;
  const float p = v * (1 - s);
  const float q = v * (1 - f * s);
  const float t = v * (1 - (1 - f) * s);
  float r, g, b;
  switch (h_i) {
    case 0:
      r = v, g = t, b = p;
      break;
    case 1:
      r = q, g = v, b = p;
      break;
    case 2:
      r = p, g = v, b = t;
      break;
    case 3:
      r = p, g = q, b = v;
      break;
    case 4:
      r = t, g = p, b = v;
      break;
    case 5:
      r = v, g = p, b = q;
      break;
    default:
      r = 1, g = 1, b = 1;
      break;
  }
  return make_tuple(static_cast<uint8_t>(b * 255), static_cast<uint8_t>(g * 255),
                    static_cast<uint8_t>(r * 255));
}

std::tuple<uint8_t, uint8_t, uint8_t> random_color(int id) {
  float h_plane = ((((unsigned int)id << 2) ^ 0x937151) % 100) / 100.0f;
  float s_plane = ((((unsigned int)id << 3) ^ 0x315793) % 100) / 100.0f;
  return hsv2bgr(h_plane, s_plane, 1);
}

};  // namespace yolo